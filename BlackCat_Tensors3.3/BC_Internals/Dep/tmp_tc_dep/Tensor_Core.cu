
#include <hip/hip_runtime.h>
//
//
//
///*
// * Shape.h
// *
// *  Created on: Jan 18, 2018
// *      Author: joseph
// */
//
//#ifndef SHAPE_H_
//#define SHAPE_H_
//
//#include <vector>
//#include "../../BC_MetaTemplateFunctions/Adhoc.h"
//#include "../../BC_Expressions/BlackCat_Internal_Definitions.h"
//#include <iostream>
//namespace BC {
//
//template<int val>
//struct Rank {
//	static constexpr int value = val;
//};
//
//
//
//template<class T, class Mathlib, class inner_rank>
//struct Tensor_Core {
//
//	static constexpr int RANK = inner_rank::value;
//
//	using _shape = std::vector<int>;
//	using scalar = typename MTF::determine_scalar<T>::type;
//
//	int sz;
//	int order;
//	bool array_ownership;
//	scalar* array;
//	_shape is;
//	_shape os;
//
//		  scalar* ary() { return array; }
//	const scalar* ary() const { return array; }
//
//	operator 	   scalar*()       { return array; }
//	operator const scalar*() const { return array; }
//
//		  scalar& operator [] (int index) 		{ return array[index]; };
//	const scalar& operator [] (int index) const { return array[index]; };
//
//	Tensor_Core(const Tensor_Core& param)
//		: is(param.is), os(param.os), sz(param.sz), order(param.order), array_ownership(true) {
//		Mathlib::initialize(array, sz);
//	}
//	Tensor_Core(_shape param)
//		: is(param), order(param.size()), array_ownership(true) {
//		os = _shape(order);
//		if (order > 0) {
//			os = _shape(order);
//			sz = is[0];
//			os[0] = is[0];
//			for (int i = 1; i < order; ++i) {
//				sz *= is[i];
//				os[i] = os[i - 1] * is[i];
//			}
//		}
//		Mathlib::initialize(array, sz);
//	}
//	Tensor_Core(scalar* ary, _shape is, _shape os, int order, int size, bool ownership)
//		: array(ary), is(is), os(os), order(order), sz(size), array_ownership(ownership) {}
//	Tensor_Core(scalar* ary)
//		: array(ary), array_ownership(false), order(0), sz(1), is(0), os(0) {}
//
//	__BC_gcpu__ int rank() const { return order; }
//	__BC_gcpu__ int size() const { return sz;    }
//	__BC_gcpu__ int rows() const { return order > 0 ? is[0] : 1; }
//	__BC_gcpu__ int cols() const { return order > 1 ? is[1] : 1; }
//	__BC_gcpu__ int dimension(int i) const { return order > i ? is[i] : 1; }
//	 void printDimensions() const { for (int i = 0; i < order; ++i) { std::cout << "["<< is[i] << "]"; } std::cout << std::endl; }
//	__BC_gcpu__ void printLDDimensions() const { for (int i = 0; i < order; ++i) { std::cout << "["<< os[i] << "]"; } std::cout << std::endl; }
//
//	__BC_gcpu__ int LD_rows() const { return order > 0 ? os[0] : 1; }
//	__BC_gcpu__ int LD_cols() const { return order > 1 ? os[1] : 1; }
//	__BC_gcpu__ int LDdimension(int i) const { return order > i + 1 ? os[i] : 1; }
//
//	__BC_gcpu__ const int* InnerShape() const { return &is[0]; }
//	__BC_gcpu__ const int* OuterShape() const { return &os[0]; }
//	void print() const { Mathlib::print(array, InnerShape(),rank(), 4); }
//
//	__BC_gcpu__ const scalar* data() const { return array; }
//	__BC_gcpu__ scalar* data()  		   { return array; }
//
//	void resetShape(_shape sh)  {
//		is = sh;
//		os = _shape(order);
//		sz = is[0];
//		os[0] = is[0];
//		for (int i = 1; i < order; ++i) {
//			sz *= is[i];
//			os[i] = os[i - 1] * is[i];
//		}
//	}
//	void ChangeShapeOwnership(bool swap = false) {
//		array_ownership = swap;
//	}
//};
//
//
//
//}
//
//#endif /* SHAPE_H_ */
//
