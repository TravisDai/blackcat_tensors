#ifdef __HIPCC__
#ifndef MATHEMATICS_GPU_H_
#define MATHEMATICS_GPU_H_

#include <hipblas.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

#include "Print.h"
#include "GPU_Implementation/GPU_impl.cu"

#include "GPU_Implementation/GPU_BLAS.h"
#include "GPU_Implementation/GPU_Misc.h"
#include "GPU_Implementation/GPU_Utility.h"


namespace BC {

class GPU :
	public GPU_Misc<GPU>,
	public GPU_Utility<GPU>,
	public GPU_BLAS<GPU> {
public:

	static constexpr int CUDA_BASE_THREADS = 256;

	static int blocks(int size) {
		return 1 + (int)(size / CUDA_BASE_THREADS);
	}
	static int threads(int sz = CUDA_BASE_THREADS) {
		return sz > CUDA_BASE_THREADS ? CUDA_BASE_THREADS : sz;
	}

	template<class T, class U>
	static void copy(T t, const U u, int sz) {
		gpu_impl::copy<<<blocks(sz),threads()>>>(t, u, sz);
		hipDeviceSynchronize();
	}

	template<int d>
	struct dimension {

		struct n1 {
			template<class T, class F> static void copy(T to, const F from, int sz) {	gpu_impl::copy<<<blocks(sz),threads()>>>(to, from, sz); }
			template<class T> 		   static void eval(T to) {	gpu_impl::eval<<<blocks(to.size()),threads()>>>(to); }
		};

		struct n2 {
			template<class T, class F> static void copy(T to, const F from, int sz) {	gpu_impl::copy2d<<<blocks(sz),threads()>>>(to, from); }
			template<class T> static void eval(T to) {	gpu_impl::eval2d<<<blocks(to.size()),threads()>>>(to); }
		};

		struct n3 {
			template<class T, class F> static void copy(T to, const F from, int sz) {	gpu_impl::copy3d<<<blocks(sz),threads()>>>(to, from); }
			template<class T> static void eval(T to) {	gpu_impl::eval3d<<<blocks(to.size()),threads()>>>(to); }
		};

		struct n4 {
			template<class T, class F> static void copy(T to, const F from, int sz) {	gpu_impl::copy4d<<<blocks(sz),threads()>>>(to, from); }
			template<class T> static void eval(T to) {	gpu_impl::eval4d<<<blocks(to.size()),threads()>>>(to); }
		};

		struct n5 {
			template<class T, class F> static void copy(T to, const F from, int sz) {	gpu_impl::copy5d<<<blocks(sz),threads()>>>(to, from); }
			template<class T> static void eval(T to) {	gpu_impl::eval5d<<<blocks(to.size()),threads()>>>(to); }
		};
		using run = std::conditional_t<(d <= 1), n1,
						std::conditional_t< d ==2, n2,
							std::conditional_t< d == 3, n3,
								std::conditional_t< d == 4, n4, n5>>>>;

		//These wonky specializations are essential for cuda to compile
		//Not sure why
		template<class T, class F>
		static void copy(T to, const F from) {
			run::copy(to,from, to.size());
			hipDeviceSynchronize();
		}

		template<class T, template<class...> class F, class... set>
		static void copy(T to, F<set...> from) {
			run::copy(to,from, to.size());
			hipDeviceSynchronize();
		}
		template<template<class...> class T, template<class...> class F, class... ts, class... fs>
		static void copy(T<ts...> to, F<fs...> from) {
			run::copy(to,from, to.size());
			hipDeviceSynchronize();
		}

		template<template<class...> class T, template<class...> class U, class... Ts, class... Us>
		static void eval(T<U<Us...>, Ts...> to) {
			run::eval(to);
			hipDeviceSynchronize();
		}

		template<template<class...> class T, class... ts>
		static void copy(T<ts...> to) {
			run::eval(to);
			hipDeviceSynchronize();
		}
	};

// THIS IS MANDATORY WITH CUDA COMPILATION FOR 9.1 --- THIS IS A BUG IN THE NVCC
	template<class T, template<class...> class U, class... set>
	static void copy(T t, U<set...> u, int sz) {
		gpu_impl::copy<<<blocks(sz),threads()>>>(t, u, sz);
		hipDeviceSynchronize();
	}
	template<template<class...> class T, template<class...> class U, class... set, class... set1>
	static void copy(T<set1...> t, U<set...> u, int sz) {
		gpu_impl::copy<<<blocks(sz),threads()>>>(t, u, sz);
		hipDeviceSynchronize();
	}



};

}

#endif /* MATHEMATICS_CPU_H_ */

#endif //if cudda cc defined
