#ifdef __HIPCC__
#ifndef BC_GPU_IMPL
#define BC_GPU_IMPL

#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include <limits>
#include <cstddef>
#include <type_traits>

namespace BC {
namespace gpu_impl {

template<class T, class J> __global__
static void copy(T t, const J j, int sz) {
	for (int i = 0; i < sz; ++i) {
		t[i] = j[i];
	}
}


template<typename T, typename J> __global__  static void copy2d(T t, const J j) {
		for (int n = 0; n < j.cols(); ++n)
			for (int m = 0; m < j.rows(); ++m)
				t(m,n) = j(m,n);
}
template<typename T, typename J> __global__ static void copy3d(T t, const J j) {
	for (int k = 0; k < j.dimension(2); ++k)
		for (int n = 0; n < j.cols(); ++n)
			for (int m = 0; m < j.rows(); ++m)
				t(m,n,k) = j(m,n,k);
}
template<typename T, typename J> __global__ static void copy4d(T t, const J j) {
	for (int l = 0; l < j.dimension(3); ++l)
		for (int k = 0; k < j.dimension(2); ++k)
			for (int n = 0; n < j.cols(); ++n)
				for (int m = 0; m < j.rows(); ++m)
					t(m,n,k,l) = j(m,n,k,l);
}
template<typename T, typename J> __global__ static void copy5d(T t, const J j) {
	for (int p = 0; p < j.dimension(4); ++p)
		for (int l = 0; l < j.dimension(3); ++l)
			for (int k = 0; k < j.dimension(2); ++k)
				for (int n = 0; n < j.dimension(1); ++n)
					for (int m = 0; m < j.dimension(0); ++m)
						t(m, n, k, l, p) = j(m, n, k, l, p);
}

template<typename T, typename J> __global__
static void fill(T t, const J j, int sz) {
	for (int i = 0; i < sz; ++i) {
		t[i] = j;
	}
}

template<class T, class U, class V> __global__
static void scalarMul(T* t, U* u, V* v) {
	*t = u[0] * v[0];
}

template<class T> __global__
static void zero(T t, int sz) {
	for (int i = 0; i < sz; ++i) {
		t[i] = 0;
	}
}

template<class T> __global__
static void eval(T t, int sz) {
	for (int i = 0; i < sz; ++i) {
		t[i];
	}
}

template<class T>
struct  _max {
	static constexpr T value = std::numeric_limits<T>::max();
};

template<class T, typename J> __global__
static void randomize(T t, J lower_bound, J upper_bound, int sz, int seed) {

	 hiprandState_t state;
	  hiprand_init(seed, /* the seed controls the sequence of random values that are produced */
	              seed, /* the sequence number is only important with multiple cores */
	              1, /* the offset is how much extra we advance in the sequence for each call, can be 0 */
	              &state);


	for (int i = 0; i < sz; ++i) {
		t[i] = hiprand(&state);
		t[i] /= 10000000000; //curand max value
		t[i] *= (upper_bound - lower_bound);
		t[i] += lower_bound;
	}
}

}


}

#endif
#endif //cudacc
