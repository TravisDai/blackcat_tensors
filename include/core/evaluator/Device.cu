/*  Project: BlackCat_Tensors
 *  Author: JosephJaspers
 *  Copyright 2018
 *
 * This Source Code Form is subject to the terms of the Mozilla Public
 * License, v. 2.0. If a copy of the MPL was not distributed with this
 * file, You can obtain one at http://mozilla.org/MPL/2.0/. */

#ifdef __HIPCC__
#ifndef BC_MATHEMATICS_DEVICE_H_
#define BC_MATHEMATICS_DEVICE_H_

#include <hipblas.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

#include "Device_Impl.cu"
#include <iostream>

namespace BC {
namespace evaluator {

struct Device {

	 template<int Dimensions>
	 struct nd_evaluator_func {

		struct n1 {
			template<class Expression>
			static void eval(Expression expression, hipStream_t stream=hipStream_t()) {
				gpu_impl::eval<<<blocks(expression.size()), threads(), 0, stream>>>(expression);
			}
		};
		struct n2 {
			template<class Expression>
			static void eval(Expression expression, hipStream_t stream=hipStream_t()) {
				gpu_impl::eval2d<<<blocks(expression.size()), threads(), 0, stream>>>(expression);
			}
		};
		struct n3 {
			template<class Expression>
			static void eval(Expression expression, hipStream_t stream=hipStream_t()) {
				gpu_impl::eval3d<<<blocks(expression.size()), threads(), 0, stream>>>(expression);
			}
		};
		struct n4 {
			template<class Expression>
			static void eval(Expression expression, hipStream_t stream=hipStream_t()) {
				gpu_impl::eval4d<<<blocks(expression.size()), threads(), 0, stream>>>(expression);
			}
		};
		struct n5 {
			template<class Expression>
			static void eval(Expression expression, hipStream_t stream=hipStream_t()) {
				gpu_impl::eval5d<<<blocks(expression.size()), threads(), 0, stream>>>(expression);
			}
		};

		using run = std::conditional_t<(Dimensions <= 1), n1,
						std::conditional_t<(Dimensions == 2), n2,
							std::conditional_t<(Dimensions == 3), n3,
								std::conditional_t<(Dimensions == 4), n4, n5>>>>;

		template<class Expression>
		static void eval(Expression expression) {
			run::eval(expression);
		}

		template<class Expression, class Context>
		static void eval(Expression expression, Context context) {
			if (context.is_default_stream()) {
				run::eval(expression);
			} else {
				run::eval(expression, context.get_cuda_stream());
			}
		}
	};

	template<int dimensions, class Expression>
	static void nd_evaluator(Expression expression) {
		nd_evaluator_func<dimensions>::eval(expression);
	}
	template<int dimensions, class Expression, class Context>
	static void nd_evaluator(Expression expression, Context context) {
		nd_evaluator_func<dimensions>::eval(expression, context);
	}

};

}
}

#endif /* MATHEMATICS_CPU_H_ */

#endif //if cudda cc defined
