/*
 * Device_Impl.h
 *
 *  Created on: Dec 3, 2018
 *      Author: joseph
 */

#ifdef __HIPCC__
#ifndef BC_RANDOM_DEVICE_DEVICE_IMPL_H_
#define BC_RANDOM_DEVICE_DEVICE_IMPL_H_

#include <hipblas.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>

namespace BC {
namespace random {
namespace device_impl {

static constexpr unsigned float_decimal_length = 100000;
template<class T> __global__
static void randomize(hiprandState_t* state, T t, float lower_bound, float upper_bound, int seed) {

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    for (; i < t.size(); i += blockDim.x * gridDim.x) {
    	skipahead(i, state);
        t[i] = hiprand(state) % float_decimal_length;
        t[i] /= float_decimal_length;
        t[i] *= (upper_bound - lower_bound);
        t[i] += lower_bound;
    }
}
}
}
}

#endif
#endif /* DEVICE_IMPL_H_ */
