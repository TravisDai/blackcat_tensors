/*
 * Device_Impl.h
 *
 *  Created on: Dec 3, 2018
 *      Author: joseph
 */

#ifdef __HIPCC__
#ifndef BC_RANDOM_DEVICE_DEVICE_IMPL_H_
#define BC_RANDOM_DEVICE_DEVICE_IMPL_H_

#include <hipblas.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>

namespace BC {
namespace random {
namespace device_impl {

template<class T> __global__
static void curand_initializer(hiprandState_t* state, int seed=1) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i == 0) {
	    hiprand_init(seed, /* the seed controls the sequence of random values that are produced */
	                  seed, /* the sequence number is only important with multiple cores */
	                  1, /* the offset is how much extra we advance in the sequence for each call, can be 0 */
	                  state);
    }
}

static constexpr unsigned float_decimal_length = 100000;
template<class T> __global__
static void randomize(hiprandState_t* state, T t, float lower_bound, float upper_bound, int seed) {

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    for (; i < t.size(); i += blockDim.x * gridDim.x) {
    	skipahead(i, state);
        t[i] = hiprand(state) % float_decimal_length;
        t[i] /= float_decimal_length;
        t[i] *= (upper_bound - lower_bound);
        t[i] += lower_bound;
    }

}
}
}
}

#endif
#endif /* DEVICE_IMPL_H_ */
