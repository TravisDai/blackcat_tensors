#include "hip/hip_runtime.h"
/*
 * To change this license header, choose License Headers in Project Properties.
 * To change this template file, choose Tools | Templates
 * and open the template in the editor.
 */

/*
 * File:   LAR_General.h
 * Author: joseph
 *
 * Created on July 23, 2017, 3:24 PM
 */

#include "BLACKCAT_GPU_MATHEMATICS.cuh"

//template<typename number_type>
//__global__ void GPU_MATHEMATICS::initialize(number_type*& d, unsigned sz) {
//    d = new number_type[sz];
//}
//
//template<typename number_type>
//__global__ void GPU_MATHEMATICS::destruction(number_type * d) {
//    delete [] d;
//}

template<typename number_type>
__global__ void GPU_MATHEMATICS::copy(number_type * store, const number_type * v, unsigned sz) {
    for (int i = 0; i < sz; ++i) {
        store[i] = v[i];
    }
}

template<typename number_type>
__global__ void GPU_MATHEMATICS::fill(number_type * m, number_type f, unsigned sz) {
    for (int i = 0; i < sz; ++i) {
        m[i] = f;
    }
}

template<typename number_type>
__global__ void GPU_MATHEMATICS::transpose(number_type * s, unsigned s_ld, const number_type * m, unsigned rows, unsigned cols, unsigned m_ld) {

	for (unsigned r = 0; r < rows; ++r) {
		for (unsigned c= 0; c< cols; ++c) {
			s[r * s_ld + c] = m[c * m_ld + r];
		}
	}
}

template<typename number_type>
__global__ void GPU_MATHEMATICS::max(number_type* max_val, const number_type* data, const unsigned* ranks, const unsigned* ld, unsigned order) {

	if (order == 1) {
		for (unsigned i = 0; i < ranks[order-1]; ++i) {
			if (*max_val < data[i]) {
				*max_val = data[i];
			}
		}
	} else {

		for (unsigned i = 0; i < ranks[order-1]; ++i) {
			max(max_val, &data[i * ld[order-1]], ranks, ld, order-1);
		}
	}
}

template<typename number_type>
__global__ void GPU_MATHEMATICS::min(number_type* min_val, const number_type* data, const unsigned* ranks, const unsigned* ld, unsigned order) {

	if (order == 1) {
		for (unsigned i = 0; i < ranks[order-1]; ++i) {
			if (*min_val < data[i]) {
				*min_val = data[i];
			}
		}
	} else {

		for (unsigned i = 0; i < ranks[order-1]; ++i) {
			min(min_val, &data[i * ld[order-1]], ranks, ld, order-1);
		}
	}
}

template<typename number_type>
__global__ void GPU_MATHEMATICS::max_index(number_type* max_val, unsigned* max_indexes, const number_type* data, const unsigned* ranks, const unsigned* ld, unsigned order) {

	if (order == 1) {
		for (unsigned i = 0; i < ranks[order-1]; ++i) {
			if (*max_val < data[i]) {
				*max_val = data[i];
				*max_indexes = i;
			}
		}
	} else {

		number_type curr_max;
		for (unsigned i = 0; i < ranks[order-1]; ++i) {
			curr_max = *max_val;
			max_index(max_val, max_indexes, &data[i * ld[order-1]], ranks, ld, order-1);

			//if modified -- update
			if (curr_max != *max_val) {
				max_indexes[order-1] = i;
			}
		}
	}
}


template<typename number_type>
__global__ void GPU_MATHEMATICS::min_index(number_type* min_val, unsigned* min_indexes, const number_type* data, const unsigned* ranks, const unsigned* ld, unsigned order) {

	if (order == 1) {
		for (unsigned i = 0; i < ranks[order-1]; ++i) {
			if (*min_val < data[i]) {
				*min_val = data[i];
				*min_indexes = i;
			}
		}
	} else {

		number_type curr_max;
		for (unsigned i = 0; i < ranks[order-1]; ++i) {
			curr_max = *min_val;
			min_index(min_val, min_indexes, &data[i * ld[order-1]], ranks, ld, order-1);

			//if modified -- update
			if (curr_max != *min_val) {
				min_indexes[order-1] = i;
			}
		}
	}
}



template<typename number_type>
__global__ void GPU_MATHEMATICS::randomize(number_type * m, number_type lower_bound, number_type upper_bound, unsigned sz) {
    for (int i = 0; i < sz; ++i) {
        m[i] = rand() % (unsigned)(upper_bound - lower_bound) + lower_bound;
    }
}

template<typename number_type>
__global__ void GPU_MATHEMATICS::randomize(number_type * m, unsigned* ld, unsigned* ranks, unsigned order, number_type lower_bound, number_type upper_bound) {
    --order;
    if (order == 0) {
    	randomize(m, lower_bound, upper_bound , ranks[order]);
    } else {
    	for (unsigned i = 0; i < ranks[order]; ++i) {
    		randomize(&m[ld[order] * i], ld, ranks, order, lower_bound, upper_bound);
    	}
    }
}
template<typename number_type>
__global__ void GPU_MATHEMATICS::print(const number_type* ary, unsigned sz) {
	for (unsigned i = 0; i < sz; ++i) {
		std::cout << ary[i] << " " << std::endl;
	}
}

template<typename number_type>
__global__ void GPU_MATHEMATICS::print(const number_type* ary, const unsigned* dims, const unsigned* lead_dims, unsigned index) {


//	if (index < 3) {
//		for (unsigned r = 0; r < dims[0]; ++r) {
//
//			if (r != 0)
//			std::cout << std::endl;
//
//			for (unsigned c = 0; c< dims[1]; ++c) {
//				auto str =std::to_string(ary[r + c * lead_dims[index - 1]]);
//				str = str.substr(0, str.length() < 3 ? str.length() : 3);
//				std::cout << str << " ";
//			}
//		}
//		std::cout << "]" << std::endl << std::endl;
//
//	} else {
//		std::cout << "[";
//		for (unsigned i = 0; i < dims[index - 1]; ++i) {
//			print(&ary[i * lead_dims[index - 1]], dims, lead_dims, index - 1);
//		}
//	}
}
