
#include <hip/hip_runtime.h>
//
//
///*
// * Shape.h
// *
// *  Created on: Jan 18, 2018
// *      Author: joseph
// */
//
//#ifndef SHAPE_H_
//#define SHAPE_H_
//
//#include "Tensor_Core_Slice.cu"
//#include "../../BC_MathLibraries/Mathematics_CPU.h"
//
//#include "Determiners.h"
//namespace BC {
//
//template<class T> struct Tensor_Core {
//
//	static constexpr int inner = _rankOf<T>;
//	static constexpr int outer = _rankOf<T>;
//	using Mathlib = _mathlib<T>;
//
//public:
//
//	using self = Tensor_Core<T>;
//	using dimlist = std::vector<int>;
//	using scalar = _scalar<T>;
//
//	static constexpr int RANK = _rankOf<T>;
//	static constexpr int LD_RANK = RANK;
//	static constexpr int LAST = RANK - 1;
//
//	scalar* array;
//	int* is = Mathlib::unified_initialize(is,RANK);
//	int* os = Mathlib::unified_initialize(os, RANK);
//public:
//
//	struct CORE {
//
//		CORE(scalar* a, int* i, int* o) : array(a), is(i), os(o) {}
//		__BC_gcpu__	      scalar& operator [] (int index) 		{ return array[index]; };
//		__BC_gcpu__	const scalar& operator [] (int index) const { return array[index]; };
//		scalar* array;
//		int* is;
//		int* os;
//
//		__BC_gcpu__ int rank() const { return RANK; }
//		__BC_gcpu__ int size() const { return RANK > 0 ? os[LAST] : 1;    }
//		__BC_gcpu__ int rows() const { return RANK > 0 ? is[0] : 1; }
//		__BC_gcpu__ int cols() const { return RANK > 1 ? is[1] : 1; }
//		__BC_gcpu__ int dimension(int i) const { return RANK > i ? is[i] : 1; }
//		__BC_gcpu__ int LD_rows() const { return RANK > 0 ? os[0] : 1; }
//		__BC_gcpu__ int LD_cols() const { return RANK > 1 ? os[1] : 1; }
//		__BC_gcpu__ int LDdimension(int i) const { return RANK > i + 1 ? os[i] : 1; }
//
//		const auto innerShape() const { return RANK > 0 ? (int*)is : &ONE; }
//		const auto outerShape() const { return RANK > 0 ? (int*)os : &ONE; }
//		operator 	   scalar*()       { return array; }
//		operator const scalar*() const { return array; }
//
//	};
//
//public:
//	//THIS CONVERSION IS IMPORTANT -- ESSENTIAL FOR DOTPRODUCT
//	operator 	   scalar*()       { return array; }
//	operator const scalar*() const { return array; }
////	operator 	   CORE()       { return CORE(array, is, os); }
////	operator const CORE() const { return CORE(array, is, os); }
//	__BC_gcpu__	      scalar& operator [] (int index) 		{ return array[index]; };
//	__BC_gcpu__	const scalar& operator [] (int index) const { return array[index]; };
//
//	Tensor_Core() {
//		static_assert(RANK == 0, "DEFAULT CONSTRUCTOR FOR TENSOR_CORE ONLY AVAILABLE FOR RANK == 0 (SCALAR)");
//		Mathlib::initialize(array, 1);
//	}
//
//	Tensor_Core(dimlist param) {
//
//		if (param.size() != RANK)
//			throw std::invalid_argument("dimlist- rank != TENSOR_CORE::RANK");
//
//		if (RANK > 0) {
//			CPU::copy(is, &param[0], RANK);
//
//			os[0] = is[0];
//			for (int i = 1; i < RANK; ++i) {
//				os[i] = os[i - 1] * is[i];
//			}
//		}
//		Mathlib::initialize(array, size());
//	}
//	Tensor_Core(const int* param) {
//		if (RANK > 0) {
//			CPU::copy(is, &param[0], RANK);
//
//			os[0] = is[0];
//			for (int i = 1; i < RANK; ++i) {
//				os[i] = os[i - 1] * is[i];
//			}
//		}
//		Mathlib::initialize(array, size());
//	}
////	Tensor_Core(const Tensor_Core& param) {
////		CPU::copy(is, param.is, RANK);
////		CPU::copy(os, param.os, RANK);
////			os[0] = is[0];
////			for (int i = 1; i < RANK; ++i) {
////				os[i] = os[i - 1] * is[i];
////			}
////
////		Mathlib::initialize(array, size());
////		CPU::copy(array, param.array, size());
////	}
////	Tensor_Core(Tensor_Core&& param) {
////		CPU::copy(is, param.is, RANK);
////		CPU::copy(os, param.os, RANK);
////		array = param.array;
////		param.array = nullptr;
////	}
//
//	__BC_gcpu__ int rank() const { return RANK; }
//	__BC_gcpu__ int size() const { return RANK > 0 ? os[LAST] : 1;    }
//	__BC_gcpu__ int rows() const { return RANK > 0 ? is[0] : 1; }
//	__BC_gcpu__ int cols() const { return RANK > 1 ? is[1] : 1; }
//	__BC_gcpu__ int dimension(int i) const { return RANK > i ? is[i] : 1; }
//
//	__BC_gcpu__ int LD_rows() const { return RANK > 0 ? os[0] : 1; }
//	__BC_gcpu__ int LD_cols() const { return RANK > 1 ? os[1] : 1; }
//	__BC_gcpu__ int LDdimension(int i) const { return RANK > i + 1 ? os[i] : 1; }
//
//	const auto innerShape() const { return RANK > 0 ? (int*)is : &ONE; }
//	const auto outerShape() const { return RANK > 0 ? (int*)os : &ONE; }
//
//	const scalar* core() const { return array; }
//		  scalar* core()  	   { return array; }
//
//	void print() const { Mathlib::print(array, this->innerShape(),rank(), 4); }
//
//	void printDimensions() const {
//		for (int i = 0; i < RANK; ++i) {
//			std::cout << "[" << is[i] << "]";
//		}
//		std::cout << std::endl;
//	}
//	void printLDDimensions() const {
//		for (int i = 0; i < RANK; ++i) {
//			std::cout << "[" << os[i] << "]";
//		}
//		std::cout << std::endl;
//	}
//
//	void resetShape(dimlist sh)  {
//		os[0] = sh[0];
//		is[0] = sh[0];
//		for (int i = 1; i < RANK; ++i) {
//			is[i] = sh[i];
//			os[i] = os[i - 1] * is[i];
//		}
//	}
//
//public:
//		  auto slice(int i) 	  { return Tensor_Slice<self>(&array[os[LAST - 1] * i], *this); }
//	const auto slice(int i) const { return Tensor_Slice<self>(&array[os[LAST - 1] * i], *this); }
//
//};
//}
//
//#endif /* SHAPE_H_ */
//