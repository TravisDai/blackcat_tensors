#ifdef __HIPCC__
#ifndef MATHEMATICS_GPU_H_
#define MATHEMATICS_GPU_H_

#include <hipblas.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

#include "Print.h"
#include "GPU_Implementation/GPU_impl.cu"
#include "GPU_Implementation/GPU_BLAS.h"
#include "GPU_Implementation/GPU_Misc.h"
#include "GPU_Implementation/GPU_Utility.h"
#include "GPU_Implementation/GPU_Constants.h"


#include "GPU_Implementation/GPU_Convolution.h"

namespace BC {

class GPU :
	public GPU_Misc<GPU>,
	public GPU_Utility<GPU>,
	public GPU_BLAS<GPU>,
	public GPU_Constants<GPU>,
	public GPU_Convolution<GPU> {
public:

	static constexpr int CUDA_BASE_THREADS = 256;

	static int blocks(int size) {
		return 1 + (int)(size / CUDA_BASE_THREADS);
	}
	static int threads(int sz = CUDA_BASE_THREADS) {
		return sz > CUDA_BASE_THREADS ? CUDA_BASE_THREADS : sz;
	}

	template<class T, class U>
	static void copy(T t, const U u, int sz) {
		gpu_impl::copy<<<blocks(sz),threads()>>>(t, u, sz);
		hipDeviceSynchronize();
	}

	template<int d>
	struct dimension {

		struct n1 { template<class T> static void eval(T to) { gpu_impl::eval<<<blocks(to.size()),threads()>>>(to);   }};
		struct n2 { template<class T> static void eval(T to) { gpu_impl::eval2d<<<blocks(to.size()),threads()>>>(to); }};
		struct n3 { template<class T> static void eval(T to) { gpu_impl::eval3d<<<blocks(to.size()),threads()>>>(to); }};
		struct n4 { template<class T> static void eval(T to) { gpu_impl::eval4d<<<blocks(to.size()),threads()>>>(to); }};
		struct n5 { template<class T> static void eval(T to) { gpu_impl::eval5d<<<blocks(to.size()),threads()>>>(to); }};
		using run = std::conditional_t<(d <= 1), n1,
						std::conditional_t< d ==2, n2,
							std::conditional_t< d == 3, n3,
								std::conditional_t< d == 4, n4, n5>>>>;

		//These wonky specializations are essential for cuda to compile
		//Not sure why
		template<class T>
		static void eval(T to) {
			run::eval(to);
#ifndef __BC_PARALLEL_SECTION__
			hipDeviceSynchronize();
#endif 
		}

		template<template<class...> class T, class... Ts>
		static void eval(T<Ts...> to) {
			run::eval(to);
			hipDeviceSynchronize();
		}

	};

// THIS IS MANDATORY WITH CUDA COMPILATION FOR 9.1 --- THIS IS A BUG IN THE NVCC
	template<class T, template<class...> class U, class... set>
	static void copy(T t, U<set...> u, int sz) {
		gpu_impl::copy<<<blocks(sz),threads()>>>(t, u, sz);
		hipDeviceSynchronize();
	}
	template<template<class...> class T, template<class...> class U, class... set, class... set1>
	static void copy(T<set1...> t, U<set...> u, int sz) {
		gpu_impl::copy<<<blocks(sz),threads()>>>(t, u, sz);
		hipDeviceSynchronize();
	}



};

}

#endif /* MATHEMATICS_CPU_H_ */

#endif //if cudda cc defined
